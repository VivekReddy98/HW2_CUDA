#include "hip/hip_runtime.h"
/*
vkarri vivek reddy karri
*/

#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <hip/hip_cooperative_groups.h>
#include <iostream>
#include <math.h>

// Variable and constant already defined on the lake.cu code so defined as extern.

using namespace cooperative_groups;

// namespace cg = cooperative_groups;

#define __DEBUG

#ifndef TSCALE
#define TSCALE 1.0
#endif

#ifndef VSQR
#define VSQR 1.0
#endif

extern int tpdt(double *, double, double);


/* -----------------Error Check and time Recording setup for GPU side of execution --------------------------------*/
#define CUDA_CALL( err )     __cudaSafeCall( err, __FILE__, __LINE__ )
#define CUDA_CHK_ERR() __cudaCheckError(__FILE__,__LINE__)

/**************************************
* void __cudaSafeCall(hipError_t err, const char *file, const int line)
* void __cudaCheckError(const char *file, const int line)
*
* These routines were taken from the GPU Computing SDK
* (http://developer.nvidia.com/gpu-computing-sdk) include file "cutil.h"
**************************************/

inline void __cudaSafeCall( hipError_t err, const char *file, const int line )
{
#ifdef __DEBUG

#pragma warning( push )
#pragma warning( disable: 4127 ) // Prevent warning on do-while(0);
  do
  {
    if ( hipSuccess != err )
    {
      fprintf( stderr, "cudaSafeCall() failed at %s:%i : %s\n",
              file, line, hipGetErrorString( err ) );
      exit( -1 );
    }
  } while ( 0 );
#pragma warning( pop )
#endif  // __DEBUG
  return;
}

inline void __cudaCheckError( const char *file, const int line )
{
#ifdef __DEBUG
#pragma warning( push )
#pragma warning( disable: 4127 ) // Prevent warning on do-while(0);
  do
  {
    hipError_t err = hipGetLastError();
    if ( hipSuccess != err )
    {
      fprintf( stderr, "cudaCheckError() failed at %s:%i : %s.\n",
               file, line, hipGetErrorString( err ) );
      exit( -1 );
    }
    // More careful checking. However, this will affect performance.
    // Comment if not needed.
    /*err = hipDeviceSynchronize();
    if( hipSuccess != err )
    {
      fprintf( stderr, "cudaCheckError() with sync failed at %s:%i : %s.\n",
               file, line, hipGetErrorString( err ) );
      exit( -1 );
    }*/
  } while ( 0 );
#pragma warning( pop )
#endif // __DEBUG
  return;
}
/* -----------------Error Check and time Recording setup for GPU side of execution --------------------------------*/


// Device-Specific code to compute f_pebble function.
__device__ double f_pebble(double p, double t)
{
  return -__expf(-TSCALE * t) * p;
}

// GPU - Specific evolve13pt function defined in V2, adopted for GPU accelaration on a single CPU.
// 1D Grid and 2D Block Style is used. Threads are alse defined as 2D.
__global__ void evolve13pt_gpu(double *un, double *uc, double *uo, double *pebbles, int n, double h, double t, double end_time, int* numiters, int TotalThreads){

  double dt = h / 2;

  grid_group g = this_grid();

  double *temp_d;

  int idx_p_1;

  int i_1;
  int j_1;
  int idx, blockId;

  // Loop over till time ends
  while(1) {

    blockId = blockIdx.x + blockIdx.y * gridDim.x;
    idx_p_1 = blockId * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x;

    for (; idx_p_1 < n*n ; idx_p_1 += TotalThreads) {

      i_1=idx_p_1/n;
      j_1=idx_p_1%n;
      idx = (j_1+2) + (i_1+2)*(n+4);

      if (idx >= (2*n + 2) && idx <= (((n+1)*n) + n + 1)) {

        un[idx] = 2*uc[idx] - uo[idx] + VSQR *(dt * dt) * (uc[idx-1] + uc[idx+1] + uc[idx + n + 4] + uc[idx - n - 4] +
                                                            0.25*(uc[idx - n - 5] + uc[idx - n - 3] + uc[idx + n + 3] + uc[idx + n + 5])+
                                                            0.125*(uc[idx-2] + uc[idx+2] + uc[idx - 2*(n + 4)] - uc[idx + 2*(n + 4)]) -
                                                            5.5 * uc[idx])/(h * h) + f_pebble(pebbles[idx_p_1],t);
      }
   }

   // Synchronize the entire grid
   g.sync();

    // Check and updte the time, if crosses break.
    if(t + dt > end_time) break;
    else{
      t = t + dt;
      // Pointer Switching optimization instead of copying data to and fro from CPU data to GPU data.
      temp_d = uc;
      uc = un;
      un = uo;
      uo = temp_d;
    }

    g.sync();
  }

}



void run_gpu(double *u, double *u0, double *u1, double *pebbles, int n, double h, double end_time, int nthreads)
{
  int pi=0;
  hipDevice_t dev;
  hipDeviceGet(&dev,0); // get handle to device 0
  hipDeviceGetAttribute(&pi, hipDeviceAttributeCooperativeLaunch, dev);

  if (pi == 1){
    // printf("Co-operative Launch Property is supported on this GPU\n");
  }
  else{
    // printf("Co-operative Launch Property is Not supported on this GPU\n");
    // exit(1);
  }

    hipEvent_t kstart, kstop;
    float ktime;

    /* HW2: Define your local variables here */
   int narea = (n+4) * (n+4);
   double t;
   double dt;

    t = 0.;
    // dt = h / 2.;

    double *un, *uc, *uo;
    int *numitersHost; // Host Side Data

    numitersHost = (int *)calloc(1, sizeof(int));

    un = (double*)calloc(narea, sizeof(double));
    uc = (double*)calloc(narea, sizeof(double));
    uo = (double*)calloc(narea, sizeof(double));
    //pb = (double*)calloc(n*n, sizeof(double));

    /* Set up device timers */
    CUDA_CALL(hipSetDevice(0));
    CUDA_CALL(hipEventCreate(&kstart));
    CUDA_CALL(hipEventCreate(&kstop));

    // Device Side data
    double *un_cuda,*uc_cuda,*uo_cuda, *pb;
    int *numiters;

  /* HW2: Add CUDA kernel call preperation code here */

    hipMalloc((void**)&un_cuda, (narea)*sizeof(double));
    hipMalloc((void**)&uc_cuda, (narea)*sizeof(double));
    hipMalloc((void**)&uo_cuda, (narea)*sizeof(double));
    hipMalloc((void**)&pb, (n*n)*sizeof(double));
    hipMalloc((void**)&numiters, sizeof(int));

    hipMemcpy(un_cuda, un, sizeof(double)*narea, hipMemcpyHostToDevice);
    hipMemcpy(uc_cuda, u1, sizeof(double)*narea, hipMemcpyHostToDevice);
    hipMemcpy(uo_cuda, u0, sizeof(double)*narea, hipMemcpyHostToDevice);
    hipMemcpy(pb, pebbles, sizeof(double)*(n*n), hipMemcpyHostToDevice);
    hipMemcpy(numiters, numitersHost, sizeof(int), hipMemcpyHostToDevice);

    /* HW2: Add main lake simulation loop here */

    int numblocksperSM;

    hipOccupancyMaxActiveBlocksPerMultiprocessor(&numblocksperSM, evolve13pt_gpu, nthreads*nthreads, 0);

    double blocks = sqrt(numblocksperSM);

    int BLKS_x = (int)floor(blocks);
    int BLKS_y = int(numblocksperSM/BLKS_x);

    if (n*n < BLKS_x*BLKS_y*nthreads*nthreads){
       BLKS_x = n / nthreads;
       BLKS_y = BLKS_x + n%nthreads;
    }

    int TotalThreads = BLKS_x*BLKS_y*nthreads*nthreads;

    std::cout << "\nNumber of Blocks Used: " << BLKS_x*BLKS_y << " Possible: " << numblocksperSM << std::endl;
    std::cout << "Number of Total Threads possible for concurrent execution: " << TotalThreads << std::endl;

    dim3 block_dim(nthreads, nthreads, 1);
    dim3 grid_dim(BLKS_x, BLKS_y, 1);

    void *kernelArgs[] = {
      (void *)&un_cuda,  (void *)&uc_cuda, (void *)&uo_cuda, (void *)&pb,
      (void *)&n, (void *)&h, (void *)&t,  (void *)&end_time, (void *)&numiters, (void *)&TotalThreads};

    /* Start GPU computation timer */
    CUDA_CALL(hipEventRecord(kstart, 0));

    CUDA_CALL(hipLaunchCooperativeKernel((void *)evolve13pt_gpu, grid_dim, block_dim, kernelArgs, 0, NULL));

    CUDA_CALL(hipDeviceSynchronize());

    CUDA_CALL(hipMemcpy(u, un_cuda, sizeof(double)*narea, hipMemcpyDeviceToHost));

    // CUDA_CALL(hipMemcpy(u, un_cuda, sizeof(double)*narea, hipMemcpyDeviceToHost));
    // CUDA_CALL(hipMemcpy(uc, uc_cuda, sizeof(double)*narea, hipMemcpyDeviceToHost));
	  // CUDA_CALL(hipMemcpy(uo, uo_cuda, sizeof(double)*narea, hipMemcpyDeviceToHost));
    // CUDA_CALL(hipMemcpy(numitersHost, numiters, sizeof(int), hipMemcpyDeviceToHost));

    /* Stop GPU computation timer */
    CUDA_CALL(hipEventRecord(kstop, 0));
    CUDA_CALL(hipEventSynchronize(kstop));
    CUDA_CALL(hipEventElapsedTime(&ktime, kstart, kstop));
    printf("GPU computation: %f msec\n", ktime);

    hipFree(un_cuda);
    hipFree(uc_cuda);
    hipFree(uo_cuda);
    hipFree(pb);
    hipFree(numiters);

    free(uc);
    free(uo);
    free(un);
    free(numitersHost);

    /* timer cleanup */
    CUDA_CALL(hipEventDestroy(kstart));
    CUDA_CALL(hipEventDestroy(kstop));
}
